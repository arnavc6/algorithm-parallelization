
#include <hip/hip_runtime.h>
#include <vector>
#include <math.h>
#include <chrono>

__global__ void collatz_kernel(size_t n, size_t* vals){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(size_t i = index; i < n; i += stride){
        size_t j = i + 1;
        int steps = 0;
        while(j != 1){
            steps++;
            if(j % 2 == 0){
                j /= 2;
            } else{
                j *= 3;
                j++;
            }
        }
        vals[i] = steps;
    }
}

std::pair<std::vector<size_t>, uint64_t> collatz_cuda(size_t n){
    std::vector<size_t> cmap(n);
    size_t* vals;
    hipMallocManaged(&vals, n*sizeof(size_t));
    for(int i = 0; i < n; i++){
        vals[i] = 0;
    }
    uint64_t start_time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    collatz_kernel<<<1024, 1024>>>(n, vals);
    hipDeviceSynchronize();
    uint64_t end_time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    // cudaDeviceSynchronize();
    for(int i = 0; i < n; i++){
        cmap[i] = vals[i];
    }
    hipFree(vals);
    return std::pair<std::vector<size_t>, uint64_t>{cmap, end_time - start_time};
}

__global__ void scalar_kernel(int factor, int* m, int s){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < s * s; i += stride){
        m[i] *= factor;
    }
}

__global__ void transpose_kernel(int* m, int s){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < s * s - 1; i += stride * (s + 1)){
        int c = i + 1;
        for(int r = i + s; r < s * s; r += s){
            int temp = m[r];
            m[r] = m[c];
            m[c] = temp;
            c++;
        }
    }
}

__global__ void matrix_kernel(int* product, int* m1, int* m2, int s){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < s * s; i += stride){
        product[i] = 0;
        int init_r = s * (i / s);
        int init_c = s * (i % s);
        int r = init_r;
        int c = init_c;
        while(r < init_r + s && c < init_c + s){
            product[i] += m1[r] * m2[c];
            r++;
            c++;
        }
    }
}

__global__ void add_kernel(int* m1, int* m2, int s){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < s * s; i += stride){
        m1[i] += m2[i];
    }
}

std::pair<std::vector<std::vector<int>>, uint64_t> two_mm_cuda(int alpha, int beta, std::vector<std::vector<int>> a, std::vector<std::vector<int>> b, std::vector<std::vector<int>> c, std::vector<std::vector<int>> d){
    int s = a.size();
    int* aPtr;
    int* bPtr;
    int* cPtr;
    int* dPtr;
    int* bc;
    int* abc;
    hipMallocManaged(&aPtr, s*s*sizeof(int));
    hipMallocManaged(&bPtr, s*s*sizeof(int));
    hipMallocManaged(&cPtr, s*s*sizeof(int));
    hipMallocManaged(&dPtr, s*s*sizeof(int));
    hipMallocManaged(&bc, s*s*sizeof(int));
    hipMallocManaged(&abc, s*s*sizeof(int));
    int col = 0;
    int row = 0;
    for(int i = 0; i < s * s; i++){
        aPtr[i] = a[row][col];
        bPtr[i] = b[row][col];
        cPtr[i] = c[row][col];
        dPtr[i] = d[row][col];
        col++;
        if(col >= s){
            col = 0;
            row++;
        }
    }
    uint64_t start_time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    transpose_kernel<<<1, 1>>>(cPtr, s);
    scalar_kernel<<<1024, 1024>>>(alpha, aPtr, s);
    hipDeviceSynchronize();
    for(int i = 0; i < s * s; i++){
        bc[i] = 0;
        int init_r = s * (i / s);
        int init_c = s * (i % s);
        row = init_r;
        col = init_c;
        while(row < init_r + s && col < init_c + s){
            bc[i] += bPtr[row] * cPtr[col];
            row++;
            col++;
        }
    }
    // matrix_kernel<<<1024, 1>>>(bc, bPtr, cPtr, s);
    // cudaDeviceSynchronize();
    transpose_kernel<<<1, 1>>>(bc, s);
    scalar_kernel<<<1024, 1024>>>(beta, dPtr, s);
    hipDeviceSynchronize();
    for(int i = 0; i < s * s; i++){
        abc[i] = 0;
        int init_r = s * (i / s);
        int init_c = s * (i % s);
        row = init_r;
        col = init_c;
        while(row < init_r + s && col < init_c + s){
            abc[i] += aPtr[row] * bc[col];
            row++;
            col++;
        }
    }
    // matrix_kernel<<<1024, 1024>>>(abc, aPtr, bc, s);
    // cudaDeviceSynchronize();
    add_kernel<<<1024, 1024>>>(abc, dPtr, s);
    hipDeviceSynchronize();
    uint64_t end_time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    std::vector<std::vector<int>> abcVec(s, std::vector<int>(s));
    col = 0;
    row = 0;
    for(int i = 0; i < s * s; i++){
        abcVec[row][col] = abc[i];
        col++;
        if(col >= s){
            col = 0;
            row++;
        }
    }
    hipFree(aPtr);
    hipFree(bPtr);
    hipFree(cPtr);
    hipFree(dPtr);
    hipFree(bc);
    hipFree(abc);
    return std::pair<std::vector<std::vector<int>>, uint64_t>{abcVec, end_time - start_time};
}

__device__ void mutex_lock(int *mutex) {
    while (atomicCAS(mutex, 0, 1) == 1);
}

__device__ void mutex_unlock(int *mutex) {
    atomicExch(mutex, 0);
}

__global__ void ising_kernel(int* lattice, int* lockTable, int* roworder, int* colorder, int s, int t){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // int up, left, target, right, down;
    int attempts;
    bool backoff = false;;
    for(int i = index; i < s; i += stride){
        // printf("%d\n", i); 
        int r = roworder[i];
        int c = colorder[i];
        if(r == 0){
            if(c == 0){
                attempts = 0;
                // printf("1\n");
                while(atomicCAS(&lockTable[0], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("2\n");
                while(atomicCAS(&lockTable[1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("3\n");
                while(atomicCAS(&lockTable[t - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("4\n");
                while(atomicCAS(&lockTable[t], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("5\n");
                while(atomicCAS(&lockTable[t * (t - 1)], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                int cost = 2 * lattice[0] * (lattice[1] + lattice[t - 1] + lattice[t] + lattice[t * (t - 1)]);
                if (cost < 0){
                    lattice[0] *= -1;
                }
                atomicExch(&lockTable[0], 0);
                atomicExch(&lockTable[1], 0);
                atomicExch(&lockTable[t - 1], 0);
                atomicExch(&lockTable[t], 0);
                atomicExch(&lockTable[t * (t - 1)], 0);
            } else if(c == t - 1){
                // printf("6\n");
                while(atomicCAS(&lockTable[0], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("7\n");
                while(atomicCAS(&lockTable[t - 2], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("8\n");
                while(atomicCAS(&lockTable[t - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("9\n");
                while(atomicCAS(&lockTable[2 * t - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("10\n");
                while(atomicCAS(&lockTable[t * t - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                int cost = 2 * lattice[0] * (lattice[t - 2] + lattice[t - 1] + lattice[2 * t - 1] + lattice[t * t - 1]);
                if (cost < 0){
                    lattice[t - 1] *= -1;
                }
                atomicExch(&lockTable[0], 0);
                atomicExch(&lockTable[t - 2], 0);
                atomicExch(&lockTable[t - 1], 0);
                atomicExch(&lockTable[2 * t - 1], 0);
                atomicExch(&lockTable[t * t - 1], 0);
            } else{
                // printf("11\n");
                while(atomicCAS(&lockTable[c - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("12\n");
                while(atomicCAS(&lockTable[c], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("13\n");
                while(atomicCAS(&lockTable[c + 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("14\n");
                while(atomicCAS(&lockTable[t + c], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("15\n");
                while(atomicCAS(&lockTable[t * (t - 1) + c], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                int cost = 2 * lattice[c] * (lattice[c - 1] + lattice[c + 1] + lattice[t + c] + lattice[t * (t - 1) + c]);
                if (cost < 0){
                    lattice[c] *= -1;
                }
                atomicExch(&lockTable[c - 1], 0);
                atomicExch(&lockTable[c], 0);
                atomicExch(&lockTable[c + 1], 0);
                atomicExch(&lockTable[t + c], 0);
                atomicExch(&lockTable[t * (t - 1) + c], 0);
            }
        } else if(r == t - 1){
            if(c == 0){
                // printf("16\n");
                while(atomicCAS(&lockTable[0], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("17\n");
                while(atomicCAS(&lockTable[t * (t - 2)], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("18\n");
                while(atomicCAS(&lockTable[t * (t - 1)], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("19\n");
                while(atomicCAS(&lockTable[t * (t - 1) + 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("20\n");
                while(atomicCAS(&lockTable[t * t - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                int cost = 2 * lattice[t * (t - 1)] * (lattice[0] + lattice[t * (t - 2)] + lattice[t * (t - 1) + 1] + lattice[t * t - 1]);
                if (cost < 0){
                    lattice[t * (t - 1)] *= -1;
                }
                atomicExch(&lockTable[0], 0);
                atomicExch(&lockTable[t * (t - 2)], 0);
                atomicExch(&lockTable[t * (t - 1)], 0);
                atomicExch(&lockTable[t * (t - 1) + 1], 0);
                atomicExch(&lockTable[t * t - 1], 0);
            } else if(c == t - 1){
                // printf("21\n");
                while(atomicCAS(&lockTable[t - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("22\n");
                while(atomicCAS(&lockTable[t * (t - 1) - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("23\n");
                while(atomicCAS(&lockTable[t * (t - 1)], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("24\n");
                while(atomicCAS(&lockTable[t * (t - 1) + 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("25\n");
                while(atomicCAS(&lockTable[t * t - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                int cost = 2 * lattice[t * t - 1] * (lattice[t - 1] + lattice[t * (t - 1) - 1] + lattice[t * (t - 1)] + lattice[t * (t - 1) + 1]);
                if (cost < 0){
                    lattice[t * t - 1] *= -1;
                }
                atomicExch(&lockTable[t - 1], 0);
                atomicExch(&lockTable[t * (t - 1) - 1], 0);
                atomicExch(&lockTable[t * (t - 1)], 0);
                atomicExch(&lockTable[t * (t - 1) + 1], 0);
                atomicExch(&lockTable[t * t - 1], 0);
            } else{
                // printf("26\n");
                while(atomicCAS(&lockTable[c], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("27\n");
                while(atomicCAS(&lockTable[t * (t - 2) + c], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("28\n");
                while(atomicCAS(&lockTable[t * (t - 1) + c - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("29\n");
                while(atomicCAS(&lockTable[t * (t - 1) + c], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("30\n");
                while(atomicCAS(&lockTable[t * (t - 1) + c + 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                int cost = 2 * lattice[t * (t - 1) + c] * (lattice[c] + lattice[t * (t - 2) + c] + lattice[t * (t - 1) + c - 1] + lattice[t * (t - 1) + c + 1]);
                if (cost < 0){
                    lattice[t * (t - 1) + c] *= -1;
                }
                atomicExch(&lockTable[c], 0);
                atomicExch(&lockTable[t * (t - 2) + c], 0);
                atomicExch(&lockTable[t * (t - 1) + c - 1], 0);
                atomicExch(&lockTable[t * (t - 1) + c], 0);
                atomicExch(&lockTable[t * (t - 1) + c + 1], 0);
            }
        } else{
            if(c == 0){
                // printf("31\n");
                while(atomicCAS(&lockTable[(r - 1) * t - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("32\n");
                while(atomicCAS(&lockTable[r * t], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("33\n");
                while(atomicCAS(&lockTable[r * t + 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("34\n");
                while(atomicCAS(&lockTable[(r + 1) * t - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("35\n");
                while(atomicCAS(&lockTable[(r + 1) * t], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                int cost = 2 * lattice[r * t] * (lattice[(r - 1) * t - 1] + lattice[r * t + 1] + lattice[(r + 1) * t - 1] + lattice[(r + 1) * t]);
                if (cost < 0){
                    lattice[r * t] *= -1;
                }
                atomicExch(&lockTable[(r - 1) * t - 1], 0);
                atomicExch(&lockTable[r * t], 0);
                atomicExch(&lockTable[r * t + 1], 0);
                atomicExch(&lockTable[(r + 1) * t - 1], 0);
                atomicExch(&lockTable[(r + 1) * t], 0);
            } else if(c == t - 1){
                // printf("36\n");
                while(atomicCAS(&lockTable[r * t - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("37\n");
                while(atomicCAS(&lockTable[r * t], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("38\n");
                while(atomicCAS(&lockTable[r * t + 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("39\n");
                while(atomicCAS(&lockTable[(r + 1) * t - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("40\n");
                while(atomicCAS(&lockTable[(r + 2) * t - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                int cost = 2 * lattice[r * (t + 1) - 1] * (lattice[r * t - 1] + lattice[r * t] + lattice[r * t + 1] + lattice[(r + 2) * t - 1]);
                if (cost < 0){
                    lattice[(r + 1) * t - 1] *= -1;
                }
                atomicExch(&lockTable[r * t - 1], 0);
                atomicExch(&lockTable[r * t], 0);
                atomicExch(&lockTable[r * t + 1], 0);
                atomicExch(&lockTable[(r + 1) * t - 1], 0);
                atomicExch(&lockTable[(r + 2) * t - 1], 0);
            } else{
                // printf("41\n");
                while(atomicCAS(&lockTable[(r - 1) * t + c], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("42\n");
                while(atomicCAS(&lockTable[r * t + c - 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("43\n");
                while(atomicCAS(&lockTable[r * t + c], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("44\n");
                while(atomicCAS(&lockTable[r * t + c + 1], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                // printf("45\n");
                while(atomicCAS(&lockTable[(r + 1) * t + c], 0, 1) == 1){
                    attempts++;
                    if(attempts > 65536){
                        backoff = true;
                        break;
                    }
                } 
                if(backoff){
                    continue;
                }
                int cost = 2 * lattice[r * t + c] * (lattice[(r - 1) * t + c] + lattice[r * t + c - 1] + lattice[r * t + c + 1] + lattice[(r + 1) * t + c]);
                if (cost < 0){
                    lattice[r * t + c] *= -1;
                }
                atomicExch(&lockTable[(r - 1) * t + c], 0);
                atomicExch(&lockTable[r * t + c - 1], 0);
                atomicExch(&lockTable[r * t + c], 0);
                atomicExch(&lockTable[r * t + c + 1], 0);
                atomicExch(&lockTable[(r + 1) * t + c], 0);
            }
        }
    }
}

std::pair<std::vector<std::vector<int>>, uint64_t> ising_cuda(std::vector<std::vector<int>> lattice, std::vector<std::pair<int, int>> order){
    int s = lattice.size();
    int* latticePtr;
    int* lockTable;
    int* roworder;
    int* colorder;
    hipMallocManaged(&latticePtr, s*s*sizeof(int));
    hipMallocManaged(&lockTable, s*s*sizeof(int));
    hipMallocManaged(&roworder, order.size()*sizeof(int));
    hipMallocManaged(&colorder, order.size()*sizeof(int));
    int col = 0;
    int row = 0;
    for(int i = 0; i < s * s; i++){
        latticePtr[i] = lattice[row][col];
        col++;
        if(col >= s){
            col = 0;
            row++;
        }
    }
    for(int i = 0; i < order.size(); i++){
        roworder[i] = order[i].first;
        colorder[i] = order[i].second;
    }
    uint64_t start_time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    ising_kernel<<<1, 1024>>>(latticePtr, lockTable, roworder, colorder, order.size(), s);
    hipDeviceSynchronize();
    uint64_t end_time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    col = 0;
    row = 0;
    for(int i = 0; i < s * s; i++){
        lattice[row][col] = latticePtr[i];
        col++;
        if(col >= s){
            col = 0;
            row++;
        }
    }
    return std::pair<std::vector<std::vector<int>>, uint64_t>{lattice, end_time - start_time};
}
