
#include <hip/hip_runtime.h>
#include <vector>
#include <math.h>
#include <chrono>

std::pair<std::vector<size_t>, uint64_t> collatz_seq(size_t n){
    std::vector<size_t> cmap(n);
    uint64_t start_time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    for(size_t i = 1; i <= n; i++){
        size_t j = i;
        int steps = 0;
        while(j != 1){
            steps++;
            if(j % 2 == 0){
                j /= 2;
            } else{
                j *= 3;
                j++;
            }
        }
        cmap[i - 1] = steps;
    }
    uint64_t end_time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    return std::pair<std::vector<size_t>, uint64_t>{cmap, end_time - start_time};
}

std::pair<std::vector<std::vector<int>>, uint64_t> two_mm_seq(int alpha, int beta, std::vector<std::vector<int>> a, std::vector<std::vector<int>> b, std::vector<std::vector<int>> c, std::vector<std::vector<int>> d){
    int s = a.size();
    std::vector<std::vector<int>> bc(s, std::vector<int>(s, 0));
    std::vector<std::vector<int>> abc(s, std::vector<int>(s, 0));
    uint64_t start_time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    for(int i = 0; i < s; i++){
        for(int j = 0; j < s; j++){
            a[i][j] *= alpha;
        }
    }
    for(int i = 0; i < s; i++){
        for(int j = 0; j < s; j++){
            for(int k = 0; k < s; k++){
                bc[i][j] += b[i][k] * c[k][j];
            }
        }
    }
    for(int i = 0; i < s; i++){
        for(int j = 0; j < s; j++){
            for(int k = 0; k < s; k++){
                abc[i][j] += a[i][k] * bc[k][j];
            }
        }
    }
    for(int i = 0; i < s; i++){
        for(int j = 0; j < s; j++){
            d[i][j] *= beta;
        }
    }
    for(int i = 0; i < s; i++){
        for(int j = 0; j < s; j++){
            abc[i][j] += d[i][j];
        }
    }
    uint64_t end_time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    return std::pair<std::vector<std::vector<int>>, uint64_t>{abc, end_time - start_time};
}

std::pair<std::vector<std::vector<int>>, uint64_t> ising_seq(std::vector<std::vector<int>> lattice, std::vector<std::pair<int, int>> order){
    int s = lattice.size();
    uint64_t start_time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    for(int i = 0; i < order.size(); i++){
        int r = order[i].first;
        int c = order[i].second;
        int cost = 2 * lattice[r][c] * (lattice[r][((c-1)%s+s)%s] + lattice[r][((c+1)%s+s)%s] + lattice[((r-1)%s+s)%s][c] + lattice[((r+1)%s+s)%s][c]);
        if (cost < 0){
            lattice[r][c] *= -1;
        }
    }
    uint64_t end_time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    return std::pair<std::vector<std::vector<int>>, uint64_t>{lattice, end_time - start_time};
}